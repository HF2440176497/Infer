#include "hip/hip_runtime.h"

#include <math.h>


#include "kernel_function.h"
#include "utils.h"


bool __check_cuda_runtime(hipError_t code, const char* op, const char* file, int line) {
	if (code != hipSuccess) {
		const char* err_name = hipGetErrorName(code);
		const char* err_message = hipGetErrorString(code);
		printf("runtime error %s:%d  %s failed. \n  code = %s, message = %s\n", file, line, op, err_name, err_message);
		return false;
	}
	return true;
}

void __log_func(const char* file, int line, const char* fmt, ...) {
    va_list vl;
    va_start(vl, fmt);
    char buffer[2048];
    string filename = file_name(file, true);
    int n = snprintf(buffer, sizeof(buffer), "[%s:%d]: ", filename.c_str(), line);
    vsnprintf(buffer + n, sizeof(buffer) - n, fmt, vl);
    fprintf(stdout, "%s\n", buffer);
}

/**
 * @param src_area 像素位置数目
 * @param src_volume 包含通道数的像素数
 * @param martix dst2src 
 * @details 适用于 NCHW 格式（通道分离）
 */
__global__ 
void resize_device_kernel_batch(uint8_t* src, int src_w, int src_h, int src_area, int src_volume, 
						float* dst, int dst_w, int dst_h, int dst_area, int dst_volume, 
						int batch_size, float padding_value, utils::AffineMat matrix) {

	int dx = blockDim.x * blockIdx.x + threadIdx.x;  // 目标图像像素处理位置
	int dy = blockDim.y * blockIdx.y + threadIdx.y;  // batch 处理位置

	if (dx < dst_area && dy < batch_size) {
		int dst_y = dx / dst_w;  // 行维度
		int dst_x = dx % dst_w;  // 列维度

		float src_x = 0;
		float src_y = 0;

		affine_project_device_kernel(&matrix, dst_x, dst_y, &src_x, &src_y);

		float c0 = padding_value, c1 = padding_value, c2 = padding_value;
		if (src_x < -1 || src_x >= src_w || src_y < -1 || src_y >= src_h) {
			// skip ...
		} else {
			int y_low = floorf(src_y); 
			int x_low = floorf(src_x); 
			int y_high = y_low + 1;
			int x_high = x_low + 1;

			uint8_t const_values[] = {  // channels == 3
				(uint8_t)padding_value, 
				(uint8_t)padding_value, 
				(uint8_t)padding_value }; 

			float ly = src_y - y_low;
			float lx = src_x - x_low;
			float hy = 1 - ly;
			float hx = 1 - lx;
			float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
			uint8_t* v1 = const_values;
			uint8_t* v2 = const_values;
			uint8_t* v3 = const_values;
			uint8_t* v4 = const_values;

            if (y_low >= 0) {
                if (x_low >= 0) {
					// src_volume: 单张像素数
					v1 = src + dy * src_volume + y_low * src_w * 3 + x_low * 3;
				}
                if (x_high < src_w) {
					v2 = src + dy * src_volume + y_low * src_w * 3 + x_high * 3;
				}
            }
            if (y_high < src_h) {
                if (x_low >= 0) {
					v3 = src + dy * src_volume + y_high * src_w * 3 + x_low * 3;
				}
                if (x_high < src_w) {
					v4 = src + dy * src_volume + y_high * src_w * 3 + x_high * 3;
				}
            }
            c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
			c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
			c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
		}  // end if-else
		float* pdst = dst + dy * dst_volume + dst_y * dst_w * 3 + dst_x * 3;
		pdst[0] = c0;
		pdst[1] = c1;
		pdst[2] = c2;
	}  // end if (dx < dst_area && dy < batch_size)

}

__device__ 
void affine_project_device_kernel(utils::AffineMat* matrix, int x, int y, float* proj_x, float* proj_y) {
	*proj_x = matrix->v0 * x + matrix->v1 * y + matrix->v2;
	*proj_y = matrix->v3 * x + matrix->v4 * y + matrix->v5;
}


/**
 * 单张图片预处理
 * @details 适用于 NHWC 格式（通道交错）
 */
__global__ 
void resize_device_kernel(uint8_t* src, int src_w, int src_h, float* dst, int dst_w, int dst_h, 
						float pad_value, utils::AffineMat matrix) {
	int dx = blockDim.x * blockIdx.x + threadIdx.x;
	int dy = blockDim.y * blockIdx.y + threadIdx.y;

	if (dx >= dst_w || dy >= dst_h) {
		printf("resize_device_kernel cross the border");
		return;
	}
	int dst_x = dx;  // 列索引
	int dst_y = dy;  // 行索引

	float src_x = 0;
	float src_y = 0;

	affine_project_device_kernel(&matrix, dst_x, dst_y, &src_x, &src_y);

	float c0 = padding_value, c1 = padding_value, c2 = padding_value;
	if (src_x < -1 || src_x >= src_w || src_y < -1 || src_y >= src_h) {
		// skip ...
	} else {
		int y_low = floorf(src_y); 
		int x_low = floorf(src_x); 
		int y_high = y_low + 1;
		int x_high = x_low + 1;

		uint8_t const_values[] = {  // channels == 3
			(uint8_t)padding_value, 
			(uint8_t)padding_value, 
			(uint8_t)padding_value }; 

		float ly = src_y - y_low;
		float lx = src_x - x_low;
		float hy = 1 - ly;
		float hx = 1 - lx;
		float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
		uint8_t* v1 = const_values;
		uint8_t* v2 = const_values;
		uint8_t* v3 = const_values;
		uint8_t* v4 = const_values;

		if (y_low >= 0) {
			if (x_low >= 0) {
				// y_low lies in height
				v1 = src + y_low * src_w * 3 + x_low * 3;
			}
			if (x_high < src_w) {
				v2 = src + y_low * src_w * 3 + x_high * 3;
			}
		}
		if (y_high < src_h) {
			if (x_low >= 0) {
				v3 = src + y_high * src_w * 3 + x_low * 3;
			}
			if (x_high < src_w) {
				v4 = src + y_high * src_w * 3 + x_high * 3;
			}
		}
		c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
		c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
		c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
	}  // end if-else

	int area = dst_w * dst_h;
	float *pdst_c0 = dst + dy * dst_w + dx;
	float *pdst_c1 = pdst_c0 + area;
	float *pdst_c2 = pdst_c1 + area;
	*pdst_c0 = c0;
	*pdst_c1 = c1;
	*pdst_c2 = c2;
}
