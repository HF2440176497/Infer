

#include <tuple>

#include "kernel_function.h"
#include "utils/processor.cuh"
#include "utils/utils.h"

/**
 * @brief 分配中间变量内存
 */
Processor::Processor() {
    resize_buffer_ = std::make_shared<TRT::Tensor>(nvinfer1::DataType::kUINT8);  // cv::Mat uint8_t
    normalize_ = utils::Norm::alpha_beta(1 / 255.0f, 0.0f);
}


Processor::~Processor() {
    if (owner_stream_ && stream_) {
        CHECK(hipStreamDestroy(stream_));
    }
    owner_stream_ = false;
    stream_ = nullptr;
}

/**
 * @param image 原始图像
 * @param net_input 保存前处理结果 用于模型输入
 * @details 拷贝输入图像到 pre_buffer
 */
void Processor::pre_compute(const cv::Mat& image, std::shared_ptr<TRT::Tensor> net_input, 
                            std::shared_ptr<utils::AffineTrans> trans) {
    if (dst_h == -1 || dst_w == -1) {  // init dst scale
        dst_h = net_input->height();
        dst_w = net_input->width();
    }
    src_w = image.cols;
    src_h = image.rows;
    size_t image_size = src_w * src_h * 3;  // bytes

    resize_buffer_->resize(std::vector<int>{1, src_h, src_w, 3});  // For cv::Mat, NHWC
    resize_buffer_->to_gpu();
    resize_buffer_->copy_from_cpu(0, image.data, image_size);

    std::tuple<int, int> from{src_w, src_h};
    std::tuple<int, int> to{dst_w, dst_h};
    trans->compute(from, to);

    resize_dev(resize_buffer_, net_input, trans);  // out: CHW BGR
    channel_swap_dev(net_input, utils::ChannelsArrange::BGR);  // out: CHW RGB
    norm_dev(net_input, utils::ChannelsArrange::RGB);  // out: CHW RGB

    int64_t timestamp = utils::timestamp_ms();
    std::string filename = std::to_string(timestamp) + ".png";
    // utils::save_float_image_chw(net_input->gpu<float>(), dst_w, dst_h, filename, utils::ChannelsArrange::RGB, true);
}

/**
 * 这里是批处理 因此需要指定 ibatch
 */
void Processor::post_compute(int ibatch, std::shared_ptr<TRT::Tensor> net_output, std::shared_ptr<TRT::Tensor> post_buffer,
                             int num_bboxes, int num_classes, int output_cdim,
                             float confidence_threshold, int max_objects, std::shared_ptr<utils::AffineTrans> trans) {
    float* image_based_output = net_output->gpu<float>(ibatch);  // offset = ibatch
    float* output_array_ptr = post_buffer->gpu<float>(ibatch);

    int unit_output_size = 1 + max_objects * NUM_BOX_ELEMENT;  // pre image out size
    CHECK(hipMemsetAsync(output_array_ptr, 0, unit_output_size, stream_));  // initialize

    dim3 grid_size = CUDATools::grid_dims(num_bboxes);
    dim3 block_size = CUDATools::block_dims(num_bboxes);

    checkCudaKernel(decode_kernel<<<grid_size, block_size, 0, stream_>>>(
        image_based_output, output_array_ptr, num_bboxes, num_classes, output_cdim, confidence_threshold, max_objects,
        trans->get_d2s()));
}


/**
 * @param max_objects 检测框至多的数目
 */
void Processor::nms_decode(int ibatch, std::shared_ptr<TRT::Tensor> post_buffer, float nms_threshold, int max_objects) {
    float* parray = post_buffer->gpu<float>(ibatch);
    auto grid = CUDATools::grid_dims(max_objects);
    auto block = CUDATools::block_dims(max_objects);
    checkCudaKernel(fast_nms_kernel<<<grid, block, 0, stream_>>>(parray, max_objects, nms_threshold));
}


void Processor::set_stream(hipStream_t stream, bool owner_stream) {
    if (owner_stream_ && stream_) {
        CHECK(hipStreamDestroy(stream_));
    }
    stream_ = stream;
    owner_stream_ = owner_stream;
}

hipStream_t Processor::get_stream() {
    return stream_;
}

/**
 * 单张图片进行预处理
 */
void Processor::resize_dev(std::shared_ptr<TRT::Tensor> pre_buffer, 
                            std::shared_ptr<TRT::Tensor> net_input, 
                            std::shared_ptr<utils::AffineTrans> trans) {

    float* dst_dev = net_input->gpu<float>();
    uint8_t* src_dev = pre_buffer->gpu<uint8_t>();

    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size((dst_w + BLOCK_SIZE - 1) / BLOCK_SIZE, (dst_h + BLOCK_SIZE - 1) / BLOCK_SIZE);

    float pad_value = 114;

    resize_device_kernel <<< grid_size, block_size, 0, stream_>>> (
        src_dev, src_w, src_h, 
        dst_dev, dst_w, dst_h,
        pad_value, trans->get_d2s());
}


/**
 * 交换通道
 */
void Processor::channel_swap_dev(std::shared_ptr<TRT::Tensor> net_input, utils::ChannelsArrange order) {

    float* dst_dev = net_input->gpu<float>();

    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size((dst_w + BLOCK_SIZE - 1) / BLOCK_SIZE, (dst_h + BLOCK_SIZE - 1) / BLOCK_SIZE);

    swap_rb_channels_kernel_chw <<< grid_size, block_size, 0, stream_>>> (dst_dev, dst_w, dst_h, order);
}

/**
 * 标准化 指定三通道排列
 */
void Processor::norm_dev(std::shared_ptr<TRT::Tensor> net_input, utils::ChannelsArrange order) {
    float* dst_dev = net_input->gpu<float>();

    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size((dst_w + BLOCK_SIZE - 1) / BLOCK_SIZE, (dst_h + BLOCK_SIZE - 1) / BLOCK_SIZE);

    normalize_kernel_chw <<< grid_size, block_size, 0, stream_>>> (dst_dev, dst_w, dst_h, normalize_, order);
}
